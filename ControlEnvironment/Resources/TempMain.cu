#include "hip/hip_runtime.h"

#include "DaxDataModel.cu"

__global__ void Execute(_DaxArray* arrays, int num_arrays, int *output)
{
  int i = threadIdx.x; 
  output[i] = arrays[i].GetNumberOfTuples();
}

#include <iostream>
using namespace std;

int main()
{
  _DaxArray h_arrays[10], *d_arrays;
  int *d_output, h_output[10];
  for (int cc=0; cc < 10; cc++)
    {
    h_arrays[cc].SetNumberOfTuples(cc*2);
    }
  hipMalloc(&d_output, sizeof(int)*10);
  hipMalloc(&d_arrays, sizeof(_DaxArray)*10);
  hipMemcpy(d_arrays, h_arrays, sizeof(_DaxArray)*10, hipMemcpyHostToDevice);
  Execute<<<1, 10>>>(d_arrays, 10, d_output);
  hipMemcpy(h_output, d_output, sizeof(int)*10, hipMemcpyDeviceToHost);
  hipFree(d_output);
  hipFree(d_arrays);
  for (int cc=0; cc < 10; cc++)
    {
    cout << h_output[cc] << endl;
    }
}
