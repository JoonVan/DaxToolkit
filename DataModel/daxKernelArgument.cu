/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

=========================================================================*/
#include "daxKernelArgument.h"

//-----------------------------------------------------------------------------
daxKernelArgument::daxKernelArgument()
{
}

//-----------------------------------------------------------------------------
daxKernelArgument::~daxKernelArgument()
{
  // release all cuda-memories allocated for the arrays.
  thrust::host_vector<DaxDataArray>::iterator iter;
  thrust::host_vector<DaxDataArray> host_arrays = this->Arrays;
  for (iter = host_arrays.begin(); iter != host_arrays.end(); ++iter)
    {
    hipFree((*iter).RawData);
    }
  this->Arrays.clear();
}

//-----------------------------------------------------------------------------
const DaxKernelArgument& daxKernelArgument::Get()
{
  this->Argument.NumberOfArrays = this->Arrays.size();
  if (this->Arrays.size() > 0)
    {
    this->Argument.Arrays = thrust::raw_pointer_cast(&this->Arrays[0]);
    }
  else
    {
    this->Argument.Arrays = NULL;
    }

  this->Argument.NumberOfDatasets = this->Datasets.size();
  if (this->Datasets.size() > 0)
    {
    this->Argument.Datasets = thrust::raw_pointer_cast(&this->Datasets[0]);
    }
  else
    {
    this->Argument.Datasets = NULL;
    }
  return this->Argument;
}
