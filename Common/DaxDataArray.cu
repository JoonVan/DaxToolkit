/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

=========================================================================*/
#include "DaxDataArray.h"
#include "assert.h"

//-----------------------------------------------------------------------------
DaxDataArray DaxDataArray::CreateAndCopy(
  eType type, eDataType dataType,
  unsigned int data_size_in_bytes, void* raw_data)
{
  DaxDataArray array = Create(type, dataType, data_size_in_bytes);
  assert(array.SizeInBytes == data_size_in_bytes);
  if (data_size_in_bytes > 0)
    {
    assert(array.RawData != NULL);
    hipMemcpy(array.RawData, raw_data, data_size_in_bytes,
      hipMemcpyHostToDevice);
    }
  return array;
}

//-----------------------------------------------------------------------------
bool DaxDataArray::CopyTo(void* raw_data, unsigned int data_size_in_bytes) const
{
  assert(this->SizeInBytes >= data_size_in_bytes);
  if (data_size_in_bytes > 0)
    {
    assert(this->RawData != NULL && raw_data != NULL);
    hipMemcpy(raw_data, this->RawData, data_size_in_bytes,
      hipMemcpyDeviceToHost);
    }
  return true;
}

//-----------------------------------------------------------------------------
DaxDataArray DaxDataArray::Create(
  eType type, eDataType dataType, unsigned int data_size_in_bytes)
{
  DaxDataArray array;
  array.Type = type;
  array.DataType = dataType;
  array.SizeInBytes = data_size_in_bytes;
  array.RawData = NULL;
  if (data_size_in_bytes > 0)
    {
    hipMalloc(&array.RawData, data_size_in_bytes);
    assert(array.RawData != NULL);
    }
  return array;
}

