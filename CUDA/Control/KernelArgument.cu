/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

=========================================================================*/
#include "CUDA/Control/KernelArgument.h"

#include <assert.h>

//-----------------------------------------------------------------------------
dax::cuda::cont::KernelArgument::KernelArgument()
{
  this->DeviceDatasets = NULL;
  this->DeviceArrays = NULL;
}

//-----------------------------------------------------------------------------
dax::cuda::cont::KernelArgument::~KernelArgument()
{
  // release all cuda-memories allocated for the arrays.
  //thrust::host_vector<dax::core::DataArray>::iterator iter;
  //thrust::host_vector<dax::core::DataArray> host_arrays = this->Arrays;
  //for (iter = host_arrays.begin(); iter != host_arrays.end(); ++iter)
  //  {
  //  hipFree((*iter).RawData);
  //  }
  //this->Arrays.clear();
  using namespace std;
  cout << "Need to free cuda-memory" << endl;
}

//-----------------------------------------------------------------------------
void dax::cuda::cont::KernelArgument::SetDataSets(const std::vector<dax::core::DataSet>& datasets)
{
  this->HostDatasets = datasets;
}

//-----------------------------------------------------------------------------
void dax::cuda::cont::KernelArgument::SetArrays(const std::vector<dax::core::DataArray>& arrays)
{
  this->HostArrays = arrays;
}

//-----------------------------------------------------------------------------
void dax::cuda::cont::KernelArgument::SetArrayMap(
  const std::map<dax::cont::DataArrayPtr, int> array_map)
{
  this->ArrayMap = array_map;
}

//-----------------------------------------------------------------------------
const dax::cuda::KernelArgument& dax::cuda::cont::KernelArgument::Get()
{
  assert(this->HostArrays.size() > 0 && this->HostDatasets.size() > 0);

  hipMalloc(&this->DeviceArrays,
    sizeof(dax::core::DataArray) * this->HostArrays.size());

  hipMalloc(&this->DeviceDatasets,
    sizeof(dax::core::DataSet) * this->HostDatasets.size());

  hipMemcpy(this->DeviceArrays, &this->HostArrays[0],
    sizeof(dax::core::DataArray) * this->HostArrays.size(),
    hipMemcpyHostToDevice);

  hipMemcpy(this->DeviceDatasets, &this->HostDatasets[0],
    sizeof(dax::core::DataSet) * this->HostDatasets.size(),
    hipMemcpyHostToDevice);

  this->Argument.NumberOfDatasets = this->HostDatasets.size();
  this->Argument.NumberOfArrays = this->HostArrays.size();
  this->Argument.Arrays = this->DeviceArrays;
  this->Argument.Datasets = this->DeviceDatasets;
  return this->Argument;
}
