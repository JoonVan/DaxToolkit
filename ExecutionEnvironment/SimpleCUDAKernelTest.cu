#include "hip/hip_runtime.h"
/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

===========================================================================*/

#include <thrust/device_vector.h>
#include "DaxExecutionEnvironment.h"
#include <iostream>
using namespace std;

#define SIZE 256

__global__ void SimpleCUDAKernelTest(float3* data_array)
{
  DaxWorkMapField work;
  data_array[work.GetItem()].x = work.GetItem();
  data_array[work.GetItem()].y = 12;
  data_array[work.GetItem()].z = 13;
}

int main()
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  cout << "deviceCount: " << deviceCount << endl;
  for (int device=0; device < deviceCount; device++)
    {
    hipDevice_t cuDevice;
    hipDeviceGet(&cuDevice, device);
    int major, minor;
    hipDeviceComputeCapability(&major, &minor, cuDevice);
    cout << "Device: " << device << " = " << major << "." << minor << endl;
    }

  thrust::host_vector<float3> host_vector(SIZE * SIZE * SIZE);
  thrust::device_vector<float3> device_vector;

  device_vector = host_vector;
  SimpleCUDAKernelTest<<<SIZE * SIZE * SIZE / 128, 128>>> ( thrust::raw_pointer_cast(
    device_vector.data()));
  host_vector = device_vector;
  for (int cc=0; cc < SIZE*SIZE*SIZE; cc++)
    {
    if (host_vector[cc].x != cc)
      {
      cout << "Mismatch at " << cc << " = " << host_vector[cc].x << endl;
      abort();
      }
    }
  return 0;
}
