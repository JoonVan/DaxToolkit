#include "hip/hip_runtime.h"
/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

===========================================================================*/
#include "DaxExecutionEnvironment.h"
#include "DaxCell.cu"

DAX_WORKLET void PointDataToCellData(DAX_IN DaxWorkMapCell& work,
  DAX_IN DaxFieldPoint& point_attribute,
  DAX_OUT DaxFieldCell& cell_attribute)
{
  DaxVector3 center = make_DaxVector3(0.5, 0.5, 0.5);
  DaxCell cell(work);
  DaxScalar scalar = cell.Interpolate(center, point_attribute, 0);
  cell_attribute.Set(work, scalar);
}

DAX_WORKLET void CellGradient(DAX_IN DaxWorkMapCell& work,
  DAX_IN DaxFieldCoordinates points,
  DAX_IN DaxFieldPoint point_attribute,
  DAX_OUT DaxFieldCell& cell_attribute)
{
  DaxScalar scalar = work.GetItem();
  DaxVector3 vec = make_DaxVector3(scalar, scalar, scalar);
  cell_attribute.Set(work, vec);
}

__global__ void Execute(DaxDataObject input_do, DaxDataObject output_p2c,
  DaxArray output_cg)
{
  DaxWorkMapCell work(input_do.CellArray);
  DaxFieldPoint in_point_scalars(input_do.PointData);
  DaxFieldCell out_cell_scalars(output_p2c.CellData);
  PointDataToCellData(work, in_point_scalars, out_cell_scalars);

  DaxFieldCoordinates in_points(input_do.PointCoordinates);
  DaxFieldCell out_cell_scalars_cg(output_cg);
  CellGradient(work, in_points, in_point_scalars, out_cell_scalars_cg);
}

#include <iostream>
using namespace std;
#define POINT_EXTENT 4
#define CELL_EXTENT 3
int main()
{
  DaxArrayIrregular point_scalars;
  point_scalars.SetNumberOfTuples(POINT_EXTENT*POINT_EXTENT*POINT_EXTENT);
  point_scalars.SetNumberOfComponents(1);
  point_scalars.Allocate();
  int cc=0;
  for (int z=0; z < POINT_EXTENT; z++)
    {
    for (int y=0; y < POINT_EXTENT; y++)
      {
      for (int x=0; x < POINT_EXTENT; x++)
        {
        point_scalars.SetValue(cc, 0, cc);
        cc++;
        }
      }
    }

  DaxArrayIrregular cell_scalars_p2c;
  cell_scalars_p2c.SetNumberOfTuples(CELL_EXTENT*CELL_EXTENT*CELL_EXTENT);
  cell_scalars_p2c.SetNumberOfComponents(1);
  cell_scalars_p2c.Allocate();
  for (int cc=0; cc < CELL_EXTENT*CELL_EXTENT*CELL_EXTENT; cc++)
    {
    cell_scalars_p2c.SetValue(cc, 0, -1);
    }

  DaxArrayIrregular cell_scalars_cg;
  cell_scalars_cg.SetNumberOfTuples(CELL_EXTENT*CELL_EXTENT*CELL_EXTENT);
  cell_scalars_cg.SetNumberOfComponents(3);
  cell_scalars_cg.Allocate();
  for (int cc=0; cc < CELL_EXTENT*CELL_EXTENT*CELL_EXTENT; cc++)
    {
    for (int kk=0; kk < 3; kk++)
      {
      cell_scalars_cg.SetValue(cc, kk, -1);
      }
    }


  DaxArrayStructuredPoints point_coordinates;
  point_coordinates.SetExtent(0, POINT_EXTENT-1, 0, POINT_EXTENT-1, 0,
    POINT_EXTENT-1);
  point_coordinates.SetSpacing(1, 1, 1);
  point_coordinates.SetOrigin(0, 0, 0);
  point_coordinates.Allocate();

  DaxArrayStructuredConnectivity cell_array;
  cell_array.SetExtent(0, POINT_EXTENT-1, 0, POINT_EXTENT-1, 0,
    POINT_EXTENT-1);
  cell_array.SetSpacing(1, 1, 1);
  cell_array.SetOrigin(0, 0, 0);
  cell_array.Allocate();

  DaxDataObject input;
  input.PointData = point_scalars;
  input.PointCoordinates = point_coordinates;
  input.CellArray = cell_array;

  DaxDataObject output_p2c;
  output_p2c.CellData = cell_scalars_p2c;

  DaxDataObject output_cg;
  output_cg.CellData = cell_scalars_cg;

  DaxDataObjectDevice d_input; d_input.CopyFrom(input);
  DaxDataObjectDevice d_output_p2c; d_output_p2c.Allocate(output_p2c);
  DaxDataObjectDevice d_output_cg; d_output_cg.Allocate(output_cg);

  Execute<<<CELL_EXTENT, CELL_EXTENT*CELL_EXTENT>>>(d_input,
    d_output_p2c, d_output_cg.CellData);

  output_p2c.CopyFrom(d_output_p2c);
  for (int cc=0; cc < CELL_EXTENT*CELL_EXTENT*CELL_EXTENT; cc++)
    {
    cout << cell_scalars_p2c.GetValue(cc, 0) << endl;
    }

  output_cg.CopyFrom(d_output_cg);
  for (int cc=0; cc < CELL_EXTENT*CELL_EXTENT*CELL_EXTENT; cc++)
    {
    for (int kk=0; kk < 3; kk++)
      {
      cout << cell_scalars_cg.GetValue(cc, kk) << ", ";
      }
    cout << endl;
    }

  d_input.FreeMemory();
  input.FreeMemory();

  d_output_p2c.FreeMemory();
  output_p2c.FreeMemory();

  d_output_cg.FreeMemory();
  output_cg.FreeMemory();
  return 0;
}
