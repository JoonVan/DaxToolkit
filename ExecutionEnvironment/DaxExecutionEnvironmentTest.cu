#include "hip/hip_runtime.h"
/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

===========================================================================*/
#include "DaxExecutionEnvironment.h"

#include "DaxArgumentsParser.h"
#include "daxDataArrayIrregular.h"
#include "daxDataBridge.h"
#include "daxImageData.h"
#include "daxKernelArgument.h"
#include "DaxKernelArgument.h"

#include "CellAverage.worklet"
#include "CellGradient.worklet"
#include "Cosine.worklet"
#include "Elevation.worklet"
#include "PointDataToCellData.worklet"
#include "Sine.worklet"
#include "Square.worklet"

#include <boost/progress.hpp>


__global__ void ExecuteElevation(DaxKernelArgument argument,
  unsigned int number_of_threads,
  unsigned int number_of_iterations)
{
  for (unsigned int cc=0; cc < number_of_iterations; cc++)
    {
    DaxWorkMapField work(cc);
    DaxFieldCoordinates in_points(
      argument.Arrays[
      argument.Datasets[0].PointCoordinatesIndex]);
    DaxFieldPoint out_point_scalars (
      argument.Arrays[
      argument.Datasets[1].PointDataIndices[0]]);
    Elevation(work, in_points, out_point_scalars);
    }
}

__global__ void ExecutePipeline1(DaxKernelArgument argument,
  unsigned int number_of_threads,
  unsigned int number_of_iterations)
{
  for (unsigned int cc=0; cc < number_of_iterations; cc++)
    {
    DaxWorkMapCell work(
      argument.Arrays[
      argument.Datasets[0].CellArrayIndex], cc);
    if (work.GetItem() < number_of_threads)
      {
      DaxFieldCoordinates in_points(
        argument.Arrays[
        argument.Datasets[0].PointCoordinatesIndex]);
      DaxFieldPoint in_point_scalars (
        argument.Arrays[
        argument.Datasets[1].PointDataIndices[0]]);
      DaxFieldCell out_cell_vectors(
        argument.Arrays[
        argument.Datasets[2].CellDataIndices[0]]);

      CellGradient(work, in_points,
        in_point_scalars, out_cell_vectors);
      }
    }
}

__global__ void ExecutePipeline2(DaxKernelArgument argument,
  unsigned int number_of_threads,
  unsigned int number_of_iterations)
{
  for (unsigned int cc=0; cc < number_of_iterations; cc++)
    {
    DaxWorkMapCell work(
      argument.Arrays[
      argument.Datasets[0].CellArrayIndex], cc);
    if (work.GetItem() < number_of_threads)
      {
      DaxFieldCoordinates in_points(
        argument.Arrays[
        argument.Datasets[0].PointCoordinatesIndex]);
      DaxFieldPoint in_point_scalars (
        argument.Arrays[
        argument.Datasets[1].PointDataIndices[0]]);
      DaxFieldCell out_cell_vectors(
        argument.Arrays[
        argument.Datasets[2].CellDataIndices[0]]);

      CellGradient(work, in_points,
        in_point_scalars, out_cell_vectors);
      Sine(work, out_cell_vectors, out_cell_vectors);
      Square(work, out_cell_vectors, out_cell_vectors);
      Cosine(work, out_cell_vectors, out_cell_vectors);
      }
    }
}

#include <iostream>
using namespace std;

daxImageDataPtr CreateInputDataSet(int dim)
{
  daxImageDataPtr imageData(new daxImageData());
  imageData->SetExtent(0, dim-1, 0, dim-1, 0, dim-1);
  imageData->SetOrigin(0, 0, 0);
  imageData->SetSpacing(1, 1, 1);
  return imageData;
}

daxImageDataPtr CreateIntermediateDataset(int dim)
{
  daxImageDataPtr imageData(new daxImageData());
  imageData->SetExtent(0, dim-1, 0, dim-1, 0, dim-1);
  imageData->SetOrigin(0, 0, 0);
  imageData->SetSpacing(1, 1, 1);

  daxDataArrayScalarPtr point_scalars (new daxDataArrayScalar());
  point_scalars->SetName("ElevationScalars");
  point_scalars->SetNumberOfTuples(imageData->GetNumberOfPoints());
  imageData->PointData.push_back(point_scalars);
  return imageData;
}

daxImageDataPtr CreateOutputDataSet(int dim)
{
  daxImageDataPtr imageData(new daxImageData());
  imageData->SetExtent(0, dim-1, 0, dim-1, 0, dim-1);
  imageData->SetOrigin(0, 0, 0);
  imageData->SetSpacing(1, 1, 1);

  daxDataArrayVector3Ptr cell_gradients (new daxDataArrayVector3());
  cell_gradients->SetName("CellScalars");
  cell_gradients->SetNumberOfTuples(imageData->GetNumberOfCells());
  imageData->CellData.push_back(cell_gradients);

  for (int x=0 ; x < dim-1; x ++)
    {
    for (int y=0 ; y < dim-1; y ++)
      {
      for (int z=0 ; z < dim-1; z ++)
        {
        cell_gradients->Set(
          z * (dim-1) * (dim-1) + y * (dim-1) + x, make_DaxVector3(-1, 0, 0));
        }
      }
    }

  return imageData;
}



int main(int argc, char* argv[])
{
  DaxArgumentsParser parser;
  if (!parser.ParseArguments(argc, argv))
    {
    return 1;
    }

  const unsigned int MAX_SIZE = parser.GetProblemSize();
  const unsigned int MAX_WARP_SIZE = parser.GetMaxWarpSize();
  const unsigned int MAX_GRID_SIZE = parser.GetMaxGridSize();

  unsigned int number_of_threads = (MAX_SIZE-1) * (MAX_SIZE-1) * (MAX_SIZE-1);
  unsigned int threadCount = min(MAX_WARP_SIZE, number_of_threads);
  unsigned int warpCount = (number_of_threads / MAX_WARP_SIZE) +
    (((number_of_threads % MAX_WARP_SIZE) == 0)? 0 : 1);
  unsigned int blockCount = min(MAX_GRID_SIZE, max(1, warpCount));
  unsigned int iterations = ceil(warpCount * 1.0 / MAX_GRID_SIZE);
  cout << "Execute iterations="
    << iterations << " : blockCount="  << blockCount
    << ", threadCount=" << threadCount << endl;

  boost::timer timer;

  timer.restart();
  daxImageDataPtr input = CreateInputDataSet(MAX_SIZE);
  daxImageDataPtr intermediate = CreateIntermediateDataset(MAX_SIZE);
  daxImageDataPtr output = CreateOutputDataSet(MAX_SIZE);
  double init_time = timer.elapsed();

  daxDataBridge bridge;
  bridge.AddInputData(input);
  bridge.AddIntermediateData(intermediate);
  bridge.AddOutputData(output);

  timer.restart();
  daxKernelArgumentPtr arg = bridge.Upload();
  if (hipDeviceSynchronize() != hipSuccess)
    {
    abort();
    }

  double upload_time = timer.elapsed();

  timer.restart();
  ExecuteElevation<<<blockCount, threadCount>>>(arg->Get(), number_of_threads, iterations);
  if (parser.GetPipeline() == DaxArgumentsParser::CELL_GRADIENT)
    {
    cout << "Pipeline #1" << endl;
    ExecutePipeline1<<<blockCount, threadCount>>>(arg->Get(), number_of_threads, iterations);
    }
  else
    {
    cout << "Pipeline #2" << endl;
    ExecutePipeline2<<<blockCount, threadCount>>>(arg->Get(), number_of_threads, iterations);
    }
  if (hipDeviceSynchronize() != hipSuccess)
    {
    abort();
    }

  double execute_time = timer.elapsed();
  timer.restart();
  bridge.Download(arg);
  if (hipDeviceSynchronize() != hipSuccess)
    {
    abort();
    }
  double download_time = timer.elapsed();

  daxDataArrayVector3* array = dynamic_cast<
    daxDataArrayVector3*>( &(*output->CellData[0]) );
  for (size_t cc=0; cc < array->GetNumberOfTuples(); cc++)
    {
    DaxVector3 value = array->Get(cc);
    if (cc < 20)
      {
      cout << cc << " : " << value.x << ", " << value.y << ", " << value.z << endl;
      }
    if (value.x == -1 || value.x > 1) 
      {
      cout << cc << " : " << value.x << ", " << value.y << ", " << value.z << endl;
      break;
      }
    }
  cout << endl << endl << "Summary: -- " << MAX_SIZE << "^3 Dataset" << endl;
  cout << "Initialize: " << init_time << endl
       << "Upload: " << upload_time << endl
       << "Execute: " << execute_time << endl
       << "Download: " << download_time << endl;
  return 0;
}
