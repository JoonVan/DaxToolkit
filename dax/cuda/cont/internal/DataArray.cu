/*=========================================================================

  This software is distributed WITHOUT ANY WARRANTY; without even
  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
  PURPOSE.  See the above copyright notice for more information.

=========================================================================*/
#include <dax/cuda/cont/internal/DataArray.h>


#include <assert.h>

//-----------------------------------------------------------------------------
dax::internal::DataArray dax::cuda::cont::internal::CreateAndCopyToDevice(
  dax::internal::DataArray::eType type, dax::internal::DataArray::eDataType dataType,
  unsigned int data_size_in_bytes, const void* raw_data)
{
  dax::internal::DataArray cur_array =
    dax::cuda::cont::internal::CreateOnDevice(type, dataType, data_size_in_bytes);
  assert(cur_array.SizeInBytes == data_size_in_bytes);
  if (data_size_in_bytes > 0)
    {
    assert(cur_array.RawData != NULL);
    hipMemcpy(cur_array.RawData, raw_data, data_size_in_bytes,
      hipMemcpyHostToDevice);
    }
  return cur_array;
}

//-----------------------------------------------------------------------------
bool dax::cuda::cont::internal::CopyToHost(const dax::internal::DataArray& array,
  void* raw_data, unsigned int data_size_in_bytes)
{
  assert(array.SizeInBytes >= data_size_in_bytes);
  if (data_size_in_bytes > 0)
    {
    assert(array.RawData != NULL && raw_data != NULL);
    hipMemcpy(raw_data, array.RawData, data_size_in_bytes,
      hipMemcpyDeviceToHost);
    }
  return true;
}

//-----------------------------------------------------------------------------
dax::internal::DataArray dax::cuda::cont::internal::CreateOnDevice(
  dax::internal::DataArray::eType type,
  dax::internal::DataArray::eDataType dataType, unsigned int data_size_in_bytes)
{
  dax::internal::DataArray cur_array;
  cur_array.Type = type;
  cur_array.DataType = dataType;
  cur_array.SizeInBytes = data_size_in_bytes;
  cur_array.RawData = NULL;
  if (data_size_in_bytes > 0)
    {
    hipMalloc(&cur_array.RawData, data_size_in_bytes);
    assert(cur_array.RawData != NULL);
    }
  return cur_array;
}

