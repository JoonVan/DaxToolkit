#include "hip/hip_runtime.h"
//=============================================================================
//
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2012 Sandia Corporation.
//  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
//  the U.S. Government retains certain rights in this software.
//
//=============================================================================

#define DAX_DEVICE_ADAPTER DAX_DEVICE_ADAPTER_ERROR
#define BOOST_SP_DISABLE_THREADS

// Tests math functions that rely on system math functions in the Cuda runtime
// environment. Ensures that the Cuda versions of the functions are behaving
// the same as the standard C math library functions.

#include <dax/cuda/cont/DeviceAdapterCuda.h>

#include <dax/math/Compare.h>
#include <dax/math/Exp.h>
#include <dax/math/Precision.h>
#include <dax/math/Sign.h>
#include <dax/math/Trig.h>

#include <dax/exec/internal/ErrorMessageBuffer.h>

#include <dax/exec/Assert.h>

#include <dax/cuda/cont/internal/testing/Testing.h>

namespace ut_CudaMath {

#define MY_ASSERT(condition, message) \
  if (!(condition)) \
    { \
    return \
        __FILE__ ":" __DAX_ASSERT_EXEC_STRINGIFY(__LINE__) ": " message \
        " (" #condition ")"; \
    }

template<class Derived>
struct MathTestFunctor
{
  // The original implementation of these kernels just had the tests in the
  // paren operater as you would expect. However, when I modified the test
  // to work in both the control (host) and execution (device) environments,
  // the two had incompatible error reporting mechanisms.  To get arround this
  // problem, I use the paren overload in a curiously recurring template
  // pattern to call the execution-only raise error method in an execution-only
  // method and macros to throw exceptions only in the control environment.

  DAX_EXEC_EXPORT
  void operator()(dax::Id) const
  {
    // Hopefully the derived class will always return constant strings that do
    // not go out of scope. If we get back garbled error strings, this is
    // probably where it happens.
    const char *message = static_cast<const Derived*>(this)->Run();
    if (message != NULL)
      {
      this->ErrorMessage.RaiseError(message);
      }
  }

  dax::exec::internal::ErrorMessageBuffer ErrorMessage;
  DAX_CONT_EXPORT
  void SetErrorMessageBuffer(
      const dax::exec::internal::ErrorMessageBuffer &errorMessage)
  {
    this->ErrorMessage = errorMessage;
  }
};

struct TestCompareKernel : public MathTestFunctor<TestCompareKernel>
{
  DAX_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(dax::math::Min(3, 8) == 3, "Got wrong min.");
    MY_ASSERT(dax::math::Min(-0.1f, -0.7f) == -0.7f, "Got wrong min.");
    MY_ASSERT(dax::math::Max(3, 8) == 8, "Got wrong max.");
    MY_ASSERT(dax::math::Max(-0.1f, -0.7f) == -0.1f, "Got wrong max.");
    return NULL;
  }
};

struct TestExpKernel : public MathTestFunctor<TestExpKernel>
{
  DAX_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(test_equal(dax::math::Pow(0.25, 2.0), dax::Scalar(0.0625)),
              "Bad power result.");
    MY_ASSERT(test_equal(dax::math::Sqrt(3.75),
                         dax::math::Pow(3.75, 0.5)),
              "Bad sqrt result.");
    MY_ASSERT(test_equal(dax::math::RSqrt(3.75),
                         dax::math::Pow(3.75, -0.5)),
              "Bad reciprocal sqrt result.");
    MY_ASSERT(test_equal(dax::math::Cbrt(3.75),
                         dax::math::Pow(3.75, 1.0/3.0)),
              "Bad cbrt result.");
    MY_ASSERT(test_equal(dax::math::RCbrt(3.75),
                         dax::math::Pow(3.75, -1.0/3.0)),
              "Bad reciprocal cbrt result.");
    MY_ASSERT(test_equal(dax::math::Exp(3.75),
                         dax::math::Pow(2.71828183, 3.75)),
              "Bad exp result.");
    MY_ASSERT(test_equal(dax::math::Exp2(3.75),
                         dax::math::Pow(2.0, 3.75)),
              "Bad exp2 result.");
    MY_ASSERT(test_equal(dax::math::ExpM1(3.75),
                         dax::math::Pow(2.71828183, 3.75)-dax::Scalar(1)),
              "Bad expm1 result.");
    MY_ASSERT(test_equal(dax::math::Exp10(3.75),
                         dax::math::Pow(10.0, 3.75)),
              "Bad exp2 result.");
    MY_ASSERT(test_equal(dax::math::Log2(dax::Scalar(0.25)),
                         dax::Scalar(-2.0)),
              "Bad value from Log2");
    MY_ASSERT(
          test_equal(dax::math::Log2(dax::make_Vector4(0.5, 1.0, 2.0, 4.0)),
                     dax::make_Vector4(-1.0, 0.0, 1.0, 2.0)),
          "Bad value from Log2");
    MY_ASSERT(test_equal(dax::math::Log(dax::Scalar(3.75)),
                         dax::Scalar(1.321755839982319)),
              "Bad log result.");
    MY_ASSERT(test_equal(dax::math::Log10(dax::Scalar(3.75)),
                         dax::Scalar(0.574031267727719)),
              "Bad log10 result.");
    MY_ASSERT(test_equal(dax::math::Log1P(3.75),
                         dax::math::Log(4.75)),
              "Bad log1p result.");
    return NULL;
  }
};

struct TestPrecisionKernel : public MathTestFunctor<TestPrecisionKernel>
{
  DAX_EXEC_CONT_EXPORT const char *Run() const
  {
    dax::Scalar zero = 0.0;
    dax::Scalar finite = 1.0;
    dax::Scalar nan = dax::math::Nan();
    dax::Scalar inf = dax::math::Infinity();
    dax::Scalar neginf = dax::math::NegativeInfinity();
    dax::Scalar epsilon = dax::math::Epsilon();

    // General behavior.
    MY_ASSERT(nan != nan, "Nan not equal itself.");
    MY_ASSERT(!(nan >= zero), "Nan not greater or less.");
    MY_ASSERT(!(nan <= zero), "Nan not greater or less.");
    MY_ASSERT(!(nan >= finite), "Nan not greater or less.");
    MY_ASSERT(!(nan <= finite), "Nan not greater or less.");

    MY_ASSERT(neginf < inf, "Infinity big");
    MY_ASSERT(zero < inf, "Infinity big");
    MY_ASSERT(finite < inf, "Infinity big");
    MY_ASSERT(zero > neginf, "-Infinity small");
    MY_ASSERT(finite > neginf, "-Infinity small");

    MY_ASSERT(zero < epsilon, "Negative epsilon");
    MY_ASSERT(finite > epsilon, "Large epsilon");

    // Math check functions.
    MY_ASSERT(!dax::math::IsNan(zero), "Bad IsNan check.");
    MY_ASSERT(!dax::math::IsNan(finite), "Bad IsNan check.");
    MY_ASSERT(dax::math::IsNan(nan), "Bad IsNan check.");
    MY_ASSERT(!dax::math::IsNan(inf), "Bad IsNan check.");
    MY_ASSERT(!dax::math::IsNan(neginf), "Bad IsNan check.");
    MY_ASSERT(!dax::math::IsNan(epsilon), "Bad IsNan check.");

    MY_ASSERT(!dax::math::IsInf(zero), "Bad infinity check.");
    MY_ASSERT(!dax::math::IsInf(finite), "Bad infinity check.");
    MY_ASSERT(!dax::math::IsInf(nan), "Bad infinity check.");
    MY_ASSERT(dax::math::IsInf(inf), "Bad infinity check.");
    MY_ASSERT(dax::math::IsInf(neginf), "Bad infinity check.");
    MY_ASSERT(!dax::math::IsInf(epsilon), "Bad infinity check.");

    MY_ASSERT(dax::math::IsFinite(zero), "Bad finite check.");
    MY_ASSERT(dax::math::IsFinite(finite), "Bad finite check.");
    MY_ASSERT(!dax::math::IsFinite(nan), "Bad finite check.");
    MY_ASSERT(!dax::math::IsFinite(inf), "Bad finite check.");
    MY_ASSERT(!dax::math::IsFinite(neginf), "Bad finite check.");
    MY_ASSERT(dax::math::IsFinite(epsilon), "Bad finite check.");

    MY_ASSERT(test_equal(dax::math::FMod(6.5, 2.3), dax::Scalar(1.9)),
              "Bad fmod.");
    MY_ASSERT(test_equal(dax::math::Remainder(6.5, 2.3),
                         dax::Scalar(-0.4)),
              "Bad remainder.");
    dax::Scalar remainder, quotient;
    remainder = dax::math::RemainderQuotient(6.5, 2.3, quotient);
    MY_ASSERT(test_equal(remainder, dax::Scalar(-0.4)), "Bad remainder.");
    MY_ASSERT(test_equal(quotient, dax::Scalar(3.0)), "Bad quotient.");
    dax::Scalar integral, fractional;
    fractional = dax::math::ModF(4.6, integral);
    MY_ASSERT(test_equal(integral, dax::Scalar(4.0)), "Bad integral.");
    MY_ASSERT(test_equal(fractional, dax::Scalar(0.6)), "Bad fractional.");
    MY_ASSERT(test_equal(dax::math::Floor(4.6), dax::Scalar(4.0)),
              "Bad floor.");
    MY_ASSERT(test_equal(dax::math::Ceil(4.6), dax::Scalar(5.0)),
              "Bad ceil.");
    MY_ASSERT(test_equal(dax::math::Round(4.6), dax::Scalar(5.0)),
              "Bad round.");

    return NULL;
  }
};

struct TestSignKernel : public MathTestFunctor<TestSignKernel>
{
  DAX_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(dax::math::Abs(-1) == 1, "Bad abs.");
    MY_ASSERT(dax::math::Abs(dax::Scalar(-0.25)) == 0.25, "Bad abs.");
    MY_ASSERT(dax::math::IsNegative(-3.1), "Bad negative.");
    MY_ASSERT(!dax::math::IsNegative(3.2), "Bad positive.");
    MY_ASSERT(!dax::math::IsNegative(0.0), "Bad non-negative.");
    MY_ASSERT(dax::math::SignBit(-3.1), "Bad negative SignBit.");
    MY_ASSERT(!dax::math::SignBit(3.2), "Bad positive SignBit.");
    MY_ASSERT(!dax::math::SignBit(0.0), "Bad non-negative SignBit.");
    MY_ASSERT(dax::math::CopySign(-0.25, 100.0) == 0.25, "Copy sign.");

    return NULL;
  }
};

struct TestTrigKernel : public MathTestFunctor<TestTrigKernel>
{
  DAX_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(test_equal(dax::math::Pi(), dax::Scalar(3.14159265)),
              "Pi not correct.");

    MY_ASSERT(test_equal(dax::math::ATan2(0.0, 1.0),
                         dax::Scalar(0.0)),
              "ATan2 x+ axis.");
    MY_ASSERT(test_equal(dax::math::ATan2(1.0, 0.0),
                         dax::Scalar(0.5*dax::math::Pi())),
              "ATan2 y+ axis.");
    MY_ASSERT(test_equal(dax::math::ATan2(-1.0, 0.0),
                         dax::Scalar(-0.5*dax::math::Pi())),
              "ATan2 y- axis.");

    MY_ASSERT(test_equal(dax::math::ATan2(1.0, 1.0),
                         dax::Scalar(0.25*dax::math::Pi())),
              "ATan2 Quadrant 1");
    MY_ASSERT(test_equal(dax::math::ATan2(1.0, -1.0),
                         dax::Scalar(0.75*dax::math::Pi())),
              "ATan2 Quadrant 2");
    MY_ASSERT(test_equal(dax::math::ATan2(-1.0, -1.0),
                         dax::Scalar(-0.75*dax::math::Pi())),
              "ATan2 Quadrant 3");
    MY_ASSERT(test_equal(dax::math::ATan2(-1.0, 1.0),
                         dax::Scalar(-0.25*dax::math::Pi())),
              "ATan2 Quadrant 4");

    dax::Scalar angle = (1.0/3.0)*dax::math::Pi();
    dax::Scalar opposite = dax::math::Sqrt(3.0);
    dax::Scalar adjacent = 1.0;
    dax::Scalar hypotenuse = 2.0;
    MY_ASSERT(test_equal(dax::math::Sin(angle), opposite/hypotenuse),
              "Sin failed test.");
    MY_ASSERT(test_equal(dax::math::Cos(angle), adjacent/hypotenuse),
              "Cos failed test.");
    MY_ASSERT(test_equal(dax::math::Tan(angle), opposite/adjacent),
              "Tan failed test.");
    MY_ASSERT(test_equal(dax::math::ASin(opposite/hypotenuse), angle),
              "Arc Sin failed test.");
    MY_ASSERT(test_equal(dax::math::ACos(adjacent/hypotenuse), angle),
              "Arc Cos failed test.");
    MY_ASSERT(test_equal(dax::math::ATan(opposite/adjacent), angle),
              "Arc Tan failed test.");

    return NULL;
  }
};

template<class Functor>
DAX_CONT_EXPORT
void TestSchedule(Functor functor)
{
  // Schedule on device.
  dax::cont::internal::DeviceAdapterAlgorithm<
      dax::cuda::cont::DeviceAdapterTagCuda>::Schedule(functor, 1);

  // Run on host. The return value has the same qualification as mentioned
  // before.
  const char *message = functor.Run();
  if (message != NULL)
    {
    DAX_TEST_FAIL(message);
    }
}

DAX_CONT_EXPORT
void TestCudaMath()
{
  std::cout << "Compare functions" << std::endl;
  TestSchedule(TestCompareKernel());

  std::cout << "Exponential functions" << std::endl;
  TestSchedule(TestExpKernel());

  std::cout << "Precision functions" << std::endl;
  TestSchedule(TestPrecisionKernel());

  std::cout << "Sign functions" << std::endl;
  TestSchedule(TestSignKernel());

  std::cout << "Trig functions" << std::endl;
  TestSchedule(TestTrigKernel());
}

} // namespace ut_CudaMath

//-----------------------------------------------------------------------------
int UnitTestCudaMath(int, char *[])
{
  return dax::cuda::cont::internal::Testing::Run(ut_CudaMath::TestCudaMath);
}
